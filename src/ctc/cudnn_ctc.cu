#include "tensors/gpu/cuda_helpers.h"
#include "ctc/cudnn_ctc.h"

namespace marian {

#ifdef CUDNN
#include <hipDNN.h>

#define CUDA_CALL(x)                  \
  do {                                \
    if((x) != hipSuccess) {          \
      printf("Error (%s) at %s:%d\n", \
             hipGetErrorString(x),   \
             __FILE__,                \
             __LINE__);               \
    }                                 \
  } while(0)


#define CUDNN_CALL(x)                 \
  do {                                \
    if((x) != HIPDNN_STATUS_SUCCESS) { \
      printf("Error (%s) at %s:%d\n", \
             hipdnnGetErrorString(x),  \
             __FILE__,                \
             __LINE__);               \
    }                                 \
  } while(0)

CUDNNCTCWrapper::CUDNNCTCWrapper(int blankTokenIndex) : CTCWrapper(blankTokenIndex) {
  CUDNN_CALL(hipdnnCreate(&cudnnHandle_));
  setCTCLossDescriptor();
}

void CUDNNCTCWrapper::setCTCLossDescriptor() {
  CUDNN_CALL(cudnnCreateCTCLossDescriptor(&ctcDesc_));
  CUDNN_CALL(cudnnSetCTCLossDescriptor_v8(ctcDesc_,
                                          HIPDNN_DATA_FLOAT,
                                          CUDNN_LOSS_NORMALIZATION_SOFTMAX,
                                          HIPDNN_NOT_PROPAGATE_NAN,
                                          256));
}

void CUDNNCTCWrapper::compute(Tensor loss,
			      Tensor grads,
			      Tensor logits,
			      Tensor flatLabels,
			      Tensor labelLengths,
			      Tensor inputLengths,
			      const Ptr<ExpressionGraph> graph) {
  CUDA_CHECK(hipSetDevice(loss->getDeviceId().no));

  Shape logitsShape = logits->shape();

  // in the first dimension is 1.
  ABORT_IF(logitsShape[0] != 1, "First dimension must be 1");
  int time = logitsShape[1];
  int batch = logitsShape[2];
  int vocab = logitsShape[3];

  // BLANK TOKEN NEEDS TO BE ON INDEX ZERO FOR CUDNN
  // need to swap values in the according columns, then move back
  float* logitsData = logits->data();

  // for every i in TIME, for every j in BATCH,

  // switch logitsData[:, :, :, blankTokenIndex_] with logitsData[:, :, :, 0].

  // axis = -1
  Tensor blankLogits;
  Tensor indices;
  Select(blankLogits, logits, indices, -1);



  const int dims[] = {time, batch, vocab};
  const int strides[] = {batch * vocab, vocab, 1};

  hipdnnTensorDescriptor_t logitsDesc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&logitsDesc));
  CUDNN_CALL(hipdnnSetTensorNdDescriptor(logitsDesc,
                                        HIPDNN_DATA_FLOAT,
                                        3,
                                        dims,
                                        strides));

  hipdnnTensorDescriptor_t gradsDesc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&gradsDesc));
  CUDNN_CALL(hipdnnSetTensorNdDescriptor(gradsDesc,
                                        HIPDNN_DATA_FLOAT,
                                        3,
                                        dims,
                                        strides));

  // TODO here, supply flat labels in CPU memory for CuDNN 7,
  // or flat labels in GPU memory for CuDNN 8.

  size_t gpuWorkspaceSize;
  CUDNN_CALL(cudnnGetCTCLossWorkspaceSize_v8(cudnnHandle_,
                                             CUDNN_CTC_LOSS_ALGO_NON_DETERMINISTIC,
                                             ctcDesc_,
                                             logitsDesc,
                                             gradsDesc,
                                             &gpuWorkspaceSize));

  //void* gpuWorkspace;
  MemoryPiece::PtrType gpuWorkspace = graph->allocator()->alloc(gpuWorkspaceSize);
  //CUDA_CHECK(hipMalloc(&gpuWorkspace, gpuWorkspaceSize));

  int* labels = flatLabels->data<int>();
  int* labelLens = labelLengths->data<int>();
  int *inputLens = inputLengths->data<int>();
  void *costs = loss->data();
  void *gradsdata = grads->data();

  hipdnnStatus_t status = cudnnCTCLoss_v8(cudnnHandle_,
                                         CUDNN_CTC_LOSS_ALGO_NON_DETERMINISTIC,
                                         ctcDesc_,
                                         logitsDesc,
                                         logitsData,
                                         labels,
                                         labelLens,
                                         inputLens,
                                         costs,
                                         gradsDesc,
                                         gradsdata,
                                         gpuWorkspaceSize,
                                         gpuWorkspace->data<void>());

  switch(status) {
    case HIPDNN_STATUS_SUCCESS:
      break;
    case HIPDNN_STATUS_BAD_PARAM:
      if (time > 256) {  // TODO not time, but label lengths!
          printf("Too many labels (%d). Limit is 256.", time);
      }
      break;
    case HIPDNN_STATUS_NOT_SUPPORTED:
    case HIPDNN_STATUS_EXECUTION_FAILED:
      printf("Error in CTC loss computation: %s", hipdnnGetErrorString(status));
      break;
  }

  //hipFree(gpuWorkspace);
  graph->allocator()->free(gpuWorkspace);

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(logitsDesc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(gradsDesc));
}

CUDNNCTCWrapper::~CUDNNCTCWrapper() {
  CUDNN_CALL(cudnnDestroyCTCLossDescriptor(ctcDesc_));
  CUDNN_CALL(hipdnnDestroy(cudnnHandle_));
}

#else  // CUDNN

CUDNNCTCWrapper::CUDNNCTCWrapper(int blankTokenIndex) {
  ABORT(
    "To use CUDNN CTC, recompile with CUDNN (cmake flag "
    "-DUSE_CUDNN=on)");
}

CUDNNCTCWrapper::~CUDNNCTCWrapper() {}

void CUDNNCTCWrapper::setCTCLossDescriptor() {
  ABORT(
    "To use CUDNN CTC, recompile with CUDNN (cmake flag "
    "-DUSE_CUDNN=on)");
}

void CUDNNCTCWrapper::compute(Tensor, Tensor, Tensor, Tensor, Tensor, Tensor,
			      const Ptr<ExpressionGraph>) {
  ABORT(
    "To use CUDNN CTC, recompile with CUDNN (cmake flag "
    "-DUSE_CUDNN=on)");
}

#endif

} // namespace marian

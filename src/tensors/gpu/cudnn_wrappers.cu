#include "hip/hip_runtime.h"
#include "tensors/gpu/cudnn_wrappers.h"

namespace marian {

#ifdef CUDNN

#include <hipDNN.h>

#define CUDA_CALL(x)                  \
  do {                                \
    if((x) != hipSuccess) {          \
      printf("Error (%s) at %s:%d\n", \
             hipGetErrorString(x),   \
             __FILE__,                \
             __LINE__);               \
    }                                 \
  } while(0)


#define CUDNN_CALL(x)                 \
  do {                                \
    if((x) != HIPDNN_STATUS_SUCCESS) { \
      printf("Error (%s) at %s:%d\n", \
             hipdnnGetErrorString(x),  \
             __FILE__,                \
             __LINE__);               \
    }                                 \
  } while(0)

CUDNNWrapper::CUDNNWrapper() {
  CUDNN_CALL(hipdnnCreate(&cudnnHandle_));
}

CUDNNWrapper::~CUDNNWrapper() {
  // std::cerr << "destroy wrapper" << std::endl;
  CUDNN_CALL(hipdnnDestroy(cudnnHandle_));
}

void CUDNNWrapper::setCudnnTensor(hipdnnTensorDescriptor_t& desc, Tensor x) {
  setCudnnTensor(desc, x->shape());
}

void CUDNNWrapper::setCudnnTensor(hipdnnTensorDescriptor_t& desc,
                                  const Shape& shape) {
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        shape[0],
                                        shape[1],
                                        shape[2],
                                        shape[3]));
}

/******************************************************************************
 * ConvolutionWrapper
 *****************************************************************************/

ConvolutionWrapper::ConvolutionWrapper(const Shape& kernelShape,
                                       const Shape& biasShape,
                                       int hPad,
                                       int wPad,
                                       int hStride,
                                       int wStride) {
  setKernelDescriptor(kernelShape);
  setConvDescriptor(hPad, wPad, hStride, wStride);
  setCudnnTensor(biasDesc_, biasShape);
}

void ConvolutionWrapper::getOutputShape(const Shape& xShape, Shape& shape) {
  hipdnnTensorDescriptor_t xDesc;
  setCudnnTensor(xDesc, xShape);
  shape.resize(4);
  CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(convDesc_,
                                                   xDesc,
                                                   kernelDesc_,
                                                   shape.data(),
                                                   shape.data() + 1,
                                                   shape.data() + 2,
                                                   shape.data() + 3));
  hipdnnDestroyTensorDescriptor(xDesc);
}

void ConvolutionWrapper::forward(Tensor x,
                                 Tensor kernels,
                                 Tensor bias,
                                 Tensor y) {
  hipSetDevice(x->getDeviceId().no);

  hipdnnTensorDescriptor_t xDesc, yDesc;
  setCudnnTensor(xDesc, x);
  setCudnnTensor(yDesc, y);

  const float alpha = 1.0f;
  const float beta = 0.0f;

  CUDNN_CALL(hipdnnConvolutionForward(cudnnHandle_,
                                     &alpha,
                                     xDesc,
                                     x->data(),
                                     kernelDesc_,
                                     kernels->data(),
                                     convDesc_,
                                     HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                     nullptr,
                                     0,
                                     &beta,
                                     yDesc,
                                     y->data()));
  CUDNN_CALL(hipdnnAddTensor(
      cudnnHandle_, &alpha, biasDesc_, bias->data(), &alpha, yDesc, y->data()));
  hipdnnDestroyTensorDescriptor(xDesc);
  hipdnnDestroyTensorDescriptor(yDesc);
}

void ConvolutionWrapper::backward(Tensor x,
                                  Tensor xGrad,
                                  Tensor kernels,
                                  Tensor kernelGrad,
                                  Tensor biasGrad,
                                  Tensor yGrad) {
  hipSetDevice(xGrad->getDeviceId().no);

  hipdnnTensorDescriptor_t xDesc, yDesc;
  setCudnnTensor(xDesc, xGrad);
  setCudnnTensor(yDesc, yGrad);

  const float alpha = 1.0f;
  const float beta = 1.0f;

  CUDNN_CALL(hipdnnConvolutionBackwardData(cudnnHandle_,
                                          &alpha,
                                          kernelDesc_,
                                          kernels->data(),
                                          yDesc,
                                          yGrad->data(),
                                          convDesc_,
                                          HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
                                          nullptr,
                                          0,
                                          &beta,
                                          xDesc,
                                          xGrad->data()));

  CUDNN_CALL(hipdnnConvolutionBackwardFilter(cudnnHandle_,
                                            &alpha,
                                            xDesc,
                                            x->data(),
                                            yDesc,
                                            yGrad->data(),
                                            convDesc_,
                                            HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
                                            nullptr,
                                            0,
                                            &beta,
                                            kernelDesc_,
                                            kernelGrad->data()));

  CUDNN_CALL(hipdnnConvolutionBackwardBias(cudnnHandle_,
                                          &alpha,
                                          yDesc,
                                          yGrad->data(),
                                          &beta,
                                          biasDesc_,
                                          biasGrad->data()));

  hipdnnDestroyTensorDescriptor(xDesc);
  hipdnnDestroyTensorDescriptor(yDesc);
}

ConvolutionWrapper::~ConvolutionWrapper() {
  // std::cerr << "destroy conv-wrapper" << std::endl;
  hipdnnDestroyConvolutionDescriptor(convDesc_);
  hipdnnDestroyFilterDescriptor(kernelDesc_);
  hipdnnDestroyTensorDescriptor(biasDesc_);
}

void ConvolutionWrapper::setConvDescriptor(int hPad,
                                           int wPad,
                                           int hStride,
                                           int wStride) {
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&convDesc_));

#if CUDNN_MAJOR > 5
  CUDNN_CALL(hipdnnSetConvolution2dDescriptor(convDesc_,
                                             hPad,
                                             wPad,
                                             hStride,
                                             wStride,
                                             1,
                                             1,  // upscales
                                             HIPDNN_CROSS_CORRELATION,
                                             HIPDNN_DATA_FLOAT));
#else
  CUDNN_CALL(hipdnnSetConvolution2dDescriptor(convDesc_,
                                             hPad,
                                             wPad,
                                             hStride,
                                             wStride,
                                             1,
                                             1,  // upscales
                                             HIPDNN_CROSS_CORRELATION));
#endif
}

void ConvolutionWrapper::setKernelDescriptor(const Shape& shape) {
  ABORT_IF(shape.size() != 4,
           "CUDN requires tensors 4D. Provided {}",
           shape.toString());
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&kernelDesc_));

  int layerIn = shape[0];
  int layerOut = shape[1];
  int kernelH = shape[2];
  int kernelW = shape[3];

  CUDNN_CALL(hipdnnSetFilter4dDescriptor(kernelDesc_,
                                        HIPDNN_DATA_FLOAT,
                                        HIPDNN_TENSOR_NCHW,
                                        layerOut,
                                        layerIn,
                                        kernelH,
                                        kernelW));
}

/******************************************************************************
 * PoolingWrapper
 *****************************************************************************/

PoolingWrapper::PoolingWrapper(int height,
                               int width,
                               int padHeight,
                               int padWidth,
                               int strideHeight,
                               int strideWidth,
                               std::string mode) {
  if(mode == "max") {
    poolingMode_ = HIPDNN_POOLING_MAX;
  } else if(mode == "avg") {
    poolingMode_ = HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;
  } else {
    ABORT("Unknown pooling mode.");
  }

  setPoolingDescriptor(
      height, width, padHeight, padWidth, strideHeight, strideWidth);
}

void PoolingWrapper::getOutputShape(const Shape& xShape, Shape& shape) {
  hipdnnTensorDescriptor_t xDesc;
  setCudnnTensor(xDesc, xShape);
  shape.resize(4);
  CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(poolingDesc_,
                                               xDesc,
                                               shape.data(),
                                               shape.data() + 1,
                                               shape.data() + 2,
                                               shape.data() + 3));
  hipdnnDestroyTensorDescriptor(xDesc);
}

void PoolingWrapper::forward(Tensor x, Tensor y) {
  hipSetDevice(x->getDeviceId().no);

  hipdnnTensorDescriptor_t xDesc, yDesc;
  setCudnnTensor(xDesc, x);
  setCudnnTensor(yDesc, y);

  const float alpha = 1.0f;
  const float beta = 0.0f;

  CUDNN_CALL(hipdnnPoolingForward(cudnnHandle_,
                                 poolingDesc_,
                                 &alpha,
                                 xDesc,
                                 x->data(),
                                 &beta,
                                 yDesc,
                                 y->data()));
  hipdnnDestroyTensorDescriptor(xDesc);
  hipdnnDestroyTensorDescriptor(yDesc);
}

void PoolingWrapper::backward(Tensor x, Tensor xGrad, Tensor y, Tensor yGrad) {
  hipSetDevice(x->getDeviceId().no);

  hipdnnTensorDescriptor_t xDesc, yDesc;
  setCudnnTensor(xDesc, x);
  setCudnnTensor(yDesc, y);

  const float alpha = 1.0f;
  const float beta = 1.0f;

  CUDNN_CALL(hipdnnPoolingBackward(cudnnHandle_,
                                  poolingDesc_,
                                  &alpha,
                                  yDesc,
                                  y->data(),
                                  yDesc,
                                  yGrad->data(),
                                  xDesc,
                                  x->data(),
                                  &beta,
                                  xDesc,
                                  xGrad->data()));
  hipdnnDestroyTensorDescriptor(xDesc);
  hipdnnDestroyTensorDescriptor(yDesc);
}

void PoolingWrapper::setPoolingDescriptor(int height,
                                          int width,
                                          int padHeight,
                                          int padWidth,
                                          int strideHeight,
                                          int strideWidth) {
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&poolingDesc_));
  CUDNN_CALL(hipdnnSetPooling2dDescriptor(poolingDesc_,
                                         poolingMode_,
                                         HIPDNN_NOT_PROPAGATE_NAN,
                                         height,
                                         width,
                                         padHeight,
                                         padWidth,
                                         strideHeight,
                                         strideWidth));
}

PoolingWrapper::~PoolingWrapper() {
  // std::cerr << "destroy pool-wrapper" << std::endl;
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(poolingDesc_));
}



/******************************************************************************
 CTCWrapper
*****************************************************************************/

CTCWrapper::CTCWrapper() {
  setCTCLossDescriptor();
}

void CTCWrapper::setCTCLossDescriptor() {
  CUDNN_CALL(cudnnCreateCTCLossDescriptor(&ctcDesc_));
  CUDNN_CALL(cudnnSetCTCLossDescriptor(ctcDesc_, HIPDNN_DATA_FLOAT));
}


void CTCWrapper::compute(Tensor logits, Tensor labels, Tensor grads) {
  hipSetDevice(grads->getDeviceId().no);

  Shape logitsShape = logits->shape();

  int time = logitsShape[0];
  int batch = logitsShape[1];
  int vocab = logitsShape[2];

  const int dims[] = {time, batch, vocab};
  const int strides[] = {batch * vocab, vocab, 1};
  
  hipdnnTensorDescriptor_t logitsDesc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&logitsDesc));
  CUDNN_CALL(hipdnnSetTensorNdDescriptor(logitsDesc,
                                        HIPDNN_DATA_FLOAT,
                                        3,
                                        dims,
                                        strides));
  
  hipdnnTensorDescriptor_t gradsDesc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&gradsDesc));
  CUDNN_CALL(hipdnnSetTensorNdDescriptor(gradsDesc,
                                        HIPDNN_DATA_FLOAT,
                                        3,
                                        dims,
                                        strides));
  
  // here, supply flat labels in CPU memory for CuDNN 7, or flat labels in GPU memory
  // for CuDNN 8.

  // label lengths
   
  size_t gpuWorkspaceSize;
  CUDNN_CALL(cudnnGetCTCWorkspaceSize(cudnnHandle_,
                                      logitsDesc,
                                      gradsDesc,
                                      labels.data(),
                                      // TODO get label lengths!, put them here
                                      // TODO get input lengths!, put them here
                                      CUDNN_CTC_LOSS_ALGO_DETERMINISTIC,
                                      ctcDesc_,
                                      &gpuWorkspaceSize));

  void* gpuWorkspace;
  CUDA_CALL(hipMalloc(&gpuWorkspace, gpuWorkspaceSize));

  float *gpuLosses;
  CUDA_CALL(hipMalloc(&gpuLosses, sizeof(float) * batch));
  // TODO losses should be output argument, no malloc here.

  hipdnnStatus_t status = cudnnCTCLoss(cudnnHandle_,
                                      logitsDesc,
                                      logits->data(),
                                      labels->data(),
                                      // TODO get label lengths!, put them here
                                      // TODO get input lengths!, put them here
                                      gpuLosses,
                                      gradsDesc,
                                      grads->data(),
                                      CUDNN_CTC_LOSS_ALGO_DETERMINISTIC,
                                      ctcDesc_,
                                      gpuWorkspace,
                                      gpuWorkspaceSize);

  switch(status) {
    case HIPDNN_STATUS_SUCCESS:
      break;
    case CUDNN_STAUS_BAD_PARAM:
      if (time > 256) {  // TODO not time, but label lengths!
          printf("Too many labels (%d). Limit is 256.", time);          
      }
      break;
    case HIPDNN_STATUS_NOT_SUPPORTED:
    case HIPDNN_STATUS_EXECUTION_FAILED:
      printf("Error in CTC loss computation: %s", hipdnnGetErrorString(status));
      break;
  }

  hipFree(gpuWorkspace);
  //TODO no free for losses here.
  
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(logitsDesc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(gradsDesc));  
}  

CTCWrapper::~CTCWrapper() {
  CUDNN_CALL(cudnnDestroyCTCLossDescriptor(ctcDesc_));
}


#else

CUDNNWrapper::CUDNNWrapper() {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

CUDNNWrapper::~CUDNNWrapper() {}

ConvolutionWrapper::ConvolutionWrapper(const Shape&,
                                       const Shape&,
                                       int,
                                       int,
                                       int,
                                       int) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

void ConvolutionWrapper::getOutputShape(const Shape&, Shape&) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

void ConvolutionWrapper::forward(Tensor, Tensor, Tensor, Tensor) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

void ConvolutionWrapper::backward(Tensor,
                                  Tensor,
                                  Tensor,
                                  Tensor,
                                  Tensor,
                                  Tensor) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

ConvolutionWrapper::~ConvolutionWrapper() {}

PoolingWrapper::PoolingWrapper(int, int, int, int, int, int, std::string) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

void PoolingWrapper::getOutputShape(const Shape&, Shape&) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

void PoolingWrapper::forward(Tensor x, Tensor y) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

void PoolingWrapper::backward(Tensor, Tensor, Tensor, Tensor) {
  ABORT(
      "To use convolution and pooling, recompile with CUDNN (cmake flag "
      "-DUSE_CUDNN=on)");
}

PoolingWrapper::~PoolingWrapper() {}

#endif
}  // namespace marian
